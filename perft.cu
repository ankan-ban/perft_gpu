#include "hip/hip_runtime.h"

//#include "chess.h"
#include "MoveGenerator088.h"
#include "MoveGeneratorBitboard.h"






class EventTimer {
public:
  EventTimer() : mStarted(false), mStopped(false) {
    hipEventCreate(&mStart);
    hipEventCreate(&mStop);
  }
  ~EventTimer() {
    hipEventDestroy(mStart);
    hipEventDestroy(mStop);
  }
  void start(hipStream_t s = 0) { hipEventRecord(mStart, s); 
                                   mStarted = true; mStopped = false; }
  void stop(hipStream_t s = 0)  { assert(mStarted);
                                   hipEventRecord(mStop, s); 
                                   mStarted = false; mStopped = true; }
  float elapsed() {
    assert(mStopped);
    if (!mStopped) return 0; 
    hipEventSynchronize(mStop);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, mStart, mStop);
    return elapsed;
  }

private:
  bool mStarted, mStopped;
  hipEvent_t mStart, mStop;
};



// for timing CPU code : start
double gTime;
#define START_TIMER { \
    LARGE_INTEGER count1, count2, freq; \
    QueryPerformanceFrequency (&freq);  \
    QueryPerformanceCounter(&count1);

#define STOP_TIMER \
    QueryPerformanceCounter(&count2); \
    gTime = ((double)(count2.QuadPart-count1.QuadPart)*1000.0)/freq.QuadPart; \
    }
// for timing CPU code : end


// perft counter function. Returns perft of the given board for given depth
uint64 perft_bb(HexaBitBoardPosition *pos, uint32 depth)
{
    HexaBitBoardPosition newPositions[256];

    /*
    if (depth == 2)
        printMoves = true;
    else
        printMoves = false;
    */

    uint32 nMoves = MoveGeneratorBitboard::generateMoves(pos, newPositions);

    if (depth == 1)
        return nMoves;

    uint64 count = 0;

    for (uint32 i=0; i < nMoves; i++)
    {
        uint64 childPerft = perft_bb(&newPositions[i], depth - 1);
        /*if (depth == 2)
            printf("%llu\n", childPerft);*/
        count += childPerft;
    }

    return count;
}



int main()
{
    BoardPosition testBoard;

    MoveGeneratorBitboard::init();

    // some test board positions from http://chessprogramming.wikispaces.com/Perft+Results

    // no bug bug till depth 7
    //Utils::readFENString("rnbqkbnr/pppppppp/8/8/8/8/PPPPPPPP/RNBQKBNR w KQkq - 0 1", &testBoard); // start.. 20 positions

    // No bug till depth 6!
    Utils::readFENString("r3k2r/p1ppqpb1/bn2pnp1/3PN3/1p2P3/2N2Q1p/PPPBBPPP/R3K2R w KQkq -", &testBoard); // position 2 (caught max bugs for me)

    // No bug till depth 7!
    // Utils::readFENString("8/2p5/3p4/KP5r/1R3p1k/8/4P1P1/8 w - -", &testBoard); // position 3

    // no bug till depth 6
    //Utils::readFENString("r2q1rk1/pP1p2pp/Q4n2/bbp1p3/Np6/1B3NBn/pPPP1PPP/R3K2R b KQ - 0 1", &testBoard); // position 4
    //Utils::readFENString("r3k2r/Pppp1ppp/1b3nbN/nP6/BBP1P3/q4N2/Pp1P2PP/R2Q1RK1 w kq - 0 1", &testBoard); // mirror of position 4
    
    // no bug till depth 6!
    //Utils::readFENString("rnbqkb1r/pp1p1ppp/2p5/4P3/2B5/8/PPP1NnPP/RNBQK2R w KQkq - 0 6", &testBoard);   // position 5

    // no bug till depth 7
    //Utils::readFENString("3Q4/1Q4Q1/4Q3/2Q4R/Q4Q2/3Q4/1Q4Rp/1K1BBNNk w - - 0 1", &testBoard); // - 218 positions.. correct!

    //Utils::readFENString("rnb1kb1r/ppqp1ppp/2p5/4P3/2B5/6K1/PPP1N1PP/RNBQ3R b kq - 0 6", &testBoard); // temp test


    HexaBitBoardPosition testBB;
    Utils::board088ToHexBB(&testBB, &testBoard);
    Utils::boardHexBBTo088(&testBoard, &testBB);

    // bug!
    printf("\nsquares between: %llu\n", MoveGeneratorBitboard::squaresInBetween(G8, B3));
    printf("\nsquares between: %llu\n", MoveGeneratorBitboard::squaresInBetween(B3, G8));

    /*
    HexaBitBoardPosition newMoves[MAX_MOVES];
    uint32 bbMoves = MoveGeneratorBitboard::generateMoves(&testBB, newMoves);
    */
    uint64 bbMoves;

    //for (int depth=1;depth<9;depth++)
    {
        int depth = 5;
        START_TIMER
        bbMoves = perft_bb(&testBB, depth);
        STOP_TIMER
        printf("\nPerft %d: %llu,   ", depth, bbMoves);
        printf("Time taken: %g seconds, nps: %llu\n", gTime/1000.0, (uint64) ((bbMoves/gTime)*1000.0));
    }
    
    //printf("\nMoves generated using bitboard: %llu\n", bbMoves);

    //printf("\nSquares in line of the given squres: %llX", MoveGeneratorBitboard::squaresInLine(C8, C4));


    //Move moves[MAX_MOVES];
    //uint32 nMoves = MoveGenerator::generateMoves(&testBoard, moves);
    //printf("\nMoves generated: %d\n", nMoves);


    printf("\nEnter FEN String: \n");
    char fen[1024];
    gets(fen);
    Utils::readFENString(fen, &testBoard); // start.. 20 positions
    Utils::dispBoard(&testBoard);

    int depth;
    printf("\nEnter depth: ");
    scanf("%d", &depth);

    //for (int depth=1;depth<7;depth++)
    {
        
        uint64 leafNodes;
        
        START_TIMER
        leafNodes = perft(&testBoard, depth);
        STOP_TIMER
        printf("\nPerft %d: %llu,   ", depth, leafNodes);
        printf("Time taken: %g seconds, nps: %llu\n", gTime/1000.0, (uint64) ((leafNodes/gTime)*1000.0));
        

#if TEST_GPU_PERFT == 1
        // try the same thing on GPU
        int hr = hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, depth);
        printf("hipDeviceSetLimit returned %d\n", hr);

        hr = hipDeviceSetLimit(hipLimitStackSize, 4*1024);
        printf("hipDeviceSetLimit stack size returned %d\n", hr);

        BoardPosition *gpuBoard;
        uint64 *gpu_perft;
        hipMalloc(&gpuBoard, sizeof(BoardPosition));
        hipMalloc(&gpu_perft, sizeof(uint64));
        hr = hipMemcpy(gpuBoard, &testBoard, sizeof(BoardPosition), hipMemcpyHostToDevice);
        printf("hipMemcpyHostToDevice returned %d\n", hr);
        EventTimer gputime;

        gputime.start();
        perft_gpu <<<1, 1>>> (gpuBoard, gpu_perft, depth, 1);
        gputime.stop();
        printf("host side launch returned: %s\n", hipGetErrorString(hipGetLastError()));

        hipDeviceSynchronize();

        uint64 res;
        hr = hipMemcpy(&res, gpu_perft, sizeof(uint64), hipMemcpyDeviceToHost);
        printf("hipMemcpyDeviceToHost returned %s\n", hipGetErrorString( (hipError_t) hr));

        printf("\nGPU Perft %d: %llu,   ", depth, res);
        printf("Time taken: %g seconds, nps: %llu\n", gputime.elapsed()/1000.0, (uint64) ((res/gputime.elapsed())*1000.0));

        hipFree(gpuBoard);
#endif
	}

    return 0;
}