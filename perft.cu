#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "perft_bb.h"
#include <math.h>
#include <stdlib.h>

//--------------------------------------------------------------------------------------------------
//  Util functions (TODO: move these to utils.cpp/.h ?)
//--------------------------------------------------------------------------------------------------

class EventTimer {
public:
  EventTimer() : mStarted(false), mStopped(false) {
    hipEventCreate(&mStart);
    hipEventCreate(&mStop);
  }
  ~EventTimer() {
    hipEventDestroy(mStart);
    hipEventDestroy(mStop);
  }
  void start(hipStream_t s = 0) { hipEventRecord(mStart, s); 
                                   mStarted = true; mStopped = false; }
  void stop(hipStream_t s = 0)  { assert(mStarted);
                                   hipEventRecord(mStop, s); 
                                   mStarted = false; mStopped = true; }
  float elapsed() {
    assert(mStopped);
    if (!mStopped) return 0; 
    hipEventSynchronize(mStop);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, mStart, mStop);
    return elapsed;
  }

private:
  bool mStarted, mStopped;
  hipEvent_t mStart, mStop;
};

// for timing CPU code : start
double gTime;
#define START_TIMER { \
    clock_t start, end; \
    start = clock();

#define STOP_TIMER \
    end = clock(); \
    gTime = (double)(end - start)/1000.0;}
// for timing CPU code : end


static void hugeMemset(void *data, uint64 size)
{
    uint8 *mem = (uint8*)data;
    const uint64 c4G = 4ull * 1024 * 1024 * 1024;

    while (size > c4G)
    {
        hipMemset(mem, 0, c4G);

        mem += c4G;
        size -= c4G;
    }

    hipMemset(mem, 0, size);
}

#if USE_TRANSPOSITION_TABLE == 1
    // can't make this bigger than 6, as the _simple kernel (breadth first search) gets called directly
    // breadth first search uses lot of memory and can can't hold bigger tree 
    #define GPU_LAUNCH_DEPTH 6
#else
    // do perft 10 using a single GPU call
    // bigger perfts are divided on the CPU
    #define GPU_LAUNCH_DEPTH 10
#endif

void allocAndClearMem(void **devPointer, void **hostPointer, size_t size, bool sysmem, int depth)
{
    hipError_t res;
    void *temp = NULL;
    *devPointer = NULL;

    if (sysmem)
    {
        if (depth >= GPU_LAUNCH_DEPTH)
        {
            // plain system memory
            temp = malloc(size);
        }
        else
        {
            // try allocating in system memory
            res = hipHostAlloc(&temp, size, hipHostMallocMapped | hipHostMallocWriteCombined);
            if (res != hipSuccess)
            {
                printf("\nFailed to allocate sysmem transposition table of %d bytes, with error: %s\n", size, hipGetErrorString(res));
            }
            res = hipHostGetDevicePointer(devPointer, temp, 0);
            if (res != S_OK)
            {
                printf("\nFailed to get GPU mapping for sysmem hash table, with error: %s\n", hipGetErrorString(res));
            }
        }
    }
    else
    {
        res = hipMalloc(devPointer, size);
        if (res != hipSuccess)
        {
            printf("\nFailed to allocate GPU transposition table of %d bytes, with error: %s\n", size, hipGetErrorString(res));
        }
    }
    *hostPointer = temp;
    if (devPointer)
    {
        hugeMemset(devPointer, size);
    }
    else
    {
        assert(*hostPointer);
        memset(*hostPointer, 0, size);
    }
}

#if USE_TRANSPOSITION_TABLE == 1
void setupHashTables128b(TTInfo128b &tt)
{
    // size of transposition tables for each depth
    // 25 bits -> 32 million entries
    // 26 bits -> 64 million ...
    //           depth->     0        1      2      3       4       5       6       7       8       9      10      11      12      13      14      15         

    const bool  shallow[] = {true, true,  true,   true,   true,  false,  false,  false,  false,  false,  false,  false,  false,  false,  false,  false};

    // settings for 12 GB card, + 16 GB sysmem
#if 0
    const uint32 ttBits[] = {0,       0,    24,     28,     27,     26,     25,     25,     25,      0,      0,      0,      0,      0,      0,      0};
    const bool   sysmem[] = {true, true, false,  false,  false,   true,   true,   true,   true,   true,   true,   true,   true,   true,   true,   true};
#else
    // settings for laptop (2 GB card + 16 GB sysmem)
    const uint32 ttBits[] = {0,       0,     25,     26,     26,     25,     25,     25,     25,      0,      0,      0,      0,      0,      0,      0};
    const bool   sysmem[] = {true, true,  false,   true,   true,   true,   true,   true,   true,   true,   true,   true,   true,   true,   true,   true};
#endif

    const int  sharedHashBits = 25;
    const bool  sharedsysmem = true;

    // allocate the shared hash table
    void *sharedTable, *sharedTableCPU;
    allocAndClearMem(&sharedTable, &sharedTableCPU, GET_TT_SIZE_FROM_BITS(sharedHashBits) * sizeof(HashEntryPerft128b), sharedsysmem, 9);

    memset(&tt, 0, sizeof(tt));
    for (int i = 2; i < MAX_PERFT_DEPTH; i++)
    {
        tt.shallowHash[i] = shallow[i];
        uint32 bits = ttBits[i];
        if (bits)
        {
            allocAndClearMem(&tt.hashTable[i], &tt.cpuTable[i],
                GET_TT_SIZE_FROM_BITS(bits) * (shallow[i] ? sizeof(HashKey128b) : sizeof(HashEntryPerft128b)), sysmem[i], i);
        }
        else
        {
            tt.hashTable[i] = sharedTable;
            tt.cpuTable[i] = sharedTableCPU;
            bits  = sharedHashBits;
        }
        tt.indexBits[i] = GET_TT_INDEX_BITS(bits);
        tt.hashBits[i] = GET_TT_HASH_BITS(bits);
    }
}
#endif

// TODO: avoid this global var?
TTInfo128b TransTables128b;

void initGPU()
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        exit(0);
    }

    // allocate the buffer to be used by device code memory allocations
    cudaStatus = hipMalloc(&preAllocatedBufferHost, PREALLOCATED_MEMORY_SIZE);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "error in malloc for preAllocatedBuffer, error desc: %s", hipGetErrorString(cudaStatus));
        exit(0);
    }
    else
    {
        printf("\nAllocated preAllocatedBuffer of %d bytes, address: %X\n", PREALLOCATED_MEMORY_SIZE, preAllocatedBufferHost);
    }

    hipMemset(&preAllocatedMemoryUsed, 0, sizeof(uint32));

#if USE_TRANSPOSITION_TABLE == 1
    setupHashTables128b(TransTables128b);
#endif
}

uint32 estimateLaunchDepth(HexaBitBoardPosition *pos)
{
    // estimate branching factor near the root
    double perft1 = perft_bb(pos, 1);
    double perft2 = perft_bb(pos, 2);
    double perft3 = perft_bb(pos, 3);

    // this works well when the root position has very low branching factor (e.g, in case king is in check)
    float geoMean = sqrt((perft3/perft2) * (perft2/perft1));
    float arithMean = ((perft3/perft2) + (perft2/perft1)) / 2;

    float branchingFactor = (geoMean + arithMean) / 2;
    if (arithMean / geoMean > 2.0f)
    {
        printf("\nUnstable position, defaulting to launch depth = 5\n");
        return 5;
    }
        
    //printf("\nEstimated branching factor: %g\n", branchingFactor);

    float memLimit = PREALLOCATED_MEMORY_SIZE / 2;  // be conservative as the branching factor can increase later

    // estimated depth is log of memLimit in base 'branchingFactor'
    uint32 depth = log(memLimit) / log (branchingFactor);

    //printf("\nEstimated launch depth: %d\n", depth);

    return depth;
}

uint64 perft_bb_cpu_launcher(HexaBitBoardPosition *pos, uint32 depth, HexaBitBoardPosition *gpuBoard, uint64 *gpu_perft, void *serial_perft_stack, int launchDepth, char *dispPrefix)
{
    HexaBitBoardPosition newPositions[MAX_MOVES];
    CMove genMoves[MAX_MOVES];
    char  dispString[128];

#if USE_TRANSPOSITION_TABLE == 1
    HashKey128b posHash128b;
    posHash128b = MoveGeneratorBitboard::computeZobristKey128b(pos);

    // check hash table
    HashEntryPerft128b *hashTable = (HashEntryPerft128b *) TransTables128b.cpuTable[depth];
    uint64 indexBits = TransTables128b.indexBits[depth];
    uint64 hashBits = TransTables128b.hashBits[depth];
    HashEntryPerft128b entry;

    if (hashTable)
    {
        entry = hashTable[posHash128b.lowPart & indexBits];
        // extract data from the entry using XORs (hash part is stored XOR'ed with data for lockless hashing scheme)
        entry.hashKey.highPart ^= entry.perftVal;
        entry.hashKey.lowPart ^= entry.perftVal;

        if ((entry.hashKey.highPart == posHash128b.highPart) && ((entry.hashKey.lowPart & hashBits) == (posHash128b.lowPart & hashBits))
            && (entry.depth == depth))
        {
            // hash hit
            return entry.perftVal;
        }
    }
#endif

    uint32 nMoves = 0;
    uint64 count = 0;

    if (depth <= GPU_LAUNCH_DEPTH)
    {
        // launch GPU perft routine
        uint64 res;
        {
            EventTimer gputime;
            gputime.start();

            hipMemcpy(gpuBoard, pos, sizeof(HexaBitBoardPosition), hipMemcpyHostToDevice);
            hipMemset(gpu_perft, 0, sizeof(uint64));
            // gpu_perft is a single 64 bit integer which is updated using atomic adds by leave nodes
            #if USE_TRANSPOSITION_TABLE == 1
                    perft_bb_gpu_simple_hash << <1, 1 >> > (gpuBoard, posHash128b, gpu_perft, depth, preAllocatedBufferHost,
                                                            TransTables128b);
            #else
                perft_bb_driver_gpu <<<1, 1>>> (gpuBoard, gpu_perft, depth, serial_perft_stack, preAllocatedBufferHost, launchDepth);
            #endif

            hipError_t err = hipMemcpy(&res, gpu_perft, sizeof(uint64), hipMemcpyDeviceToHost);
            gputime.stop();
#if USE_TRANSPOSITION_TABLE == 0
            if (err != S_OK) printf("hipMemcpyDeviceToHost returned %s\n", hipGetErrorString(err));
            printf("\nGPU Perft %d: %llu,   ", depth, res);
            fflush(stdout);
            printf("Time taken: %g seconds, nps: %llu\n", gputime.elapsed()/1000.0, (uint64) (((double) res/gputime.elapsed())*1000.0));
#endif
        }

        count = res;
    }
    else
    {
        //nMoves = generateBoards(pos, newPositions);
        // generate moves also so that we can print them

#if 1
         nMoves = generateMoves(pos, pos->chance, genMoves);
#else
        // Ankan TODO: there is some bug here!

        // generate all non-captures first to reduce hash table trashing
        if (pos->chance == WHITE)
        {
            ExpandedBitBoard ebb;
            ebb = MoveGeneratorBitboard::ExpandBitBoard<WHITE>(pos);
            nMoves  = MoveGeneratorBitboard::generateNonCaptures<WHITE>(&ebb, genMoves);
            nMoves += MoveGeneratorBitboard::generateCaptures<WHITE>(&ebb, &genMoves[nMoves]);
        }
        else
        {
            ExpandedBitBoard ebb;
            ebb = MoveGeneratorBitboard::ExpandBitBoard<BLACK>(pos);
            nMoves = MoveGeneratorBitboard::generateNonCaptures<BLACK>(&ebb, genMoves);
            nMoves += MoveGeneratorBitboard::generateCaptures<BLACK>(&ebb, &genMoves[nMoves]);
        }
#endif
        for (uint32 i = 0; i < nMoves; i++)
        {
            newPositions[i] = *pos;
            uint64 fakeHash = 0;

            if (pos->chance == WHITE)
                MoveGeneratorBitboard::makeMove<WHITE, false>(&newPositions[i], fakeHash, genMoves[i]);
            else
                MoveGeneratorBitboard::makeMove<BLACK, false>(&newPositions[i], fakeHash, genMoves[i]);

            char moveString[10];
            Utils::getCompactMoveString(genMoves[i], moveString);
            strcpy(dispString, dispPrefix);
            strcat(dispString, moveString);
            uint64 childPerft = perft_bb_cpu_launcher(&newPositions[i], depth - 1, gpuBoard, gpu_perft, serial_perft_stack, launchDepth, dispString);
            //printf("%s   %20llu\n", dispString, childPerft);
            //fflush(stdout);
            count += childPerft;
        }
    }

#if USE_TRANSPOSITION_TABLE == 1
    // store in hash table
    // replace only if old entry was shallower (or of same depth)
    if (hashTable && entry.depth <= depth)
    {
        HashEntryPerft128b newEntry;
        newEntry.perftVal = count;
        newEntry.hashKey.highPart = posHash128b.highPart;
        newEntry.hashKey.lowPart = (posHash128b.lowPart & hashBits);
        newEntry.depth = depth;

        // XOR hash part with data part for lockless hashing
        newEntry.hashKey.lowPart ^= newEntry.perftVal;
        newEntry.hashKey.highPart ^= newEntry.perftVal;

        hashTable[posHash128b.lowPart & indexBits] = newEntry;
    }
#endif
    return count;
}


// called only for bigger perfts - shows move count distribution for each move
void dividedPerft(HexaBitBoardPosition *pos, uint32 depth, int launchDepth)
{
    HexaBitBoardPosition *gpuBoard;
    uint64 *gpu_perft;
    void *serial_perft_stack;

    hipError_t cudaStatus;

    cudaStatus = hipMalloc(&gpuBoard, sizeof(HexaBitBoardPosition));
    if (cudaStatus != hipSuccess) printf("hipMalloc failed for gpuBoard, Err id: %d, str: %s\n", cudaStatus, hipGetErrorString(cudaStatus));

    cudaStatus = hipMalloc(&serial_perft_stack, GPU_SERIAL_PERFT_STACK_SIZE);
    if (cudaStatus != hipSuccess) printf("hipMalloc failed for serial_perft_stack, Err id: %d, str: %s\n", cudaStatus, hipGetErrorString(cudaStatus));

    cudaStatus = hipMalloc(&gpu_perft, sizeof(uint64));
    if (cudaStatus != hipSuccess) printf("hipMalloc failed for gpu_perft, Err id: %d, str: %s\n", cudaStatus, hipGetErrorString(cudaStatus));

    printf("\n");
    uint64 perft;
    START_TIMER
    perft = perft_bb_cpu_launcher(pos, depth, gpuBoard, gpu_perft, serial_perft_stack, launchDepth, "..");
    STOP_TIMER

#if USE_TRANSPOSITION_TABLE == 1
    printf("Perft(%02d):%20llu, time: %8g s\n", depth, perft, gTime);
    fflush(stdout);
#endif
	hipFree(gpuBoard);
    hipFree(gpu_perft);
    hipFree(serial_perft_stack);    
}

int main(int argc, char *argv[])
{
    BoardPosition testBoard;
    initGPU();
    MoveGeneratorBitboard::init();

    // some test board positions from http://chessprogramming.wikispaces.com/Perft+Results
    //Utils::readFENString("rnbqkbnr/pppppppp/8/8/8/8/PPPPPPPP/RNBQKBNR w KQkq - 0 1", &testBoard); // start.. 20 positions
    Utils::readFENString("r3k2r/p1ppqpb1/bn2pnp1/3PN3/1p2P3/2N2Q1p/PPPBBPPP/R3K2R w KQkq -", &testBoard); // position 2 (caught max bugs for me)
    //Utils::readFENString("8/2p5/3p4/KP5r/1R3p1k/8/4P1P1/8 w - -", &testBoard); // position 3
    //Utils::readFENString("r2q1rk1/pP1p2pp/Q4n2/bbp1p3/Np6/1B3NBn/pPPP1PPP/R3K2R b KQ - 0 1", &testBoard); // position 4
    //Utils::readFENString("r3k2r/Pppp1ppp/1b3nbN/nP6/BBP1P3/q4N2/Pp1P2PP/R2Q1RK1 w kq - 0 1", &testBoard); // mirror of position 4
    //Utils::readFENString("rnbqkb1r/pp1p1ppp/2p5/4P3/2B5/8/PPP1NnPP/RNBQK2R w KQkq - 0 6", &testBoard);   // position 5
    //Utils::readFENString("3Q4/1Q4Q1/4Q3/2Q4R/Q4Q2/3Q4/1Q4Rp/1K1BBNNk w - - 0 1", &testBoard); // - 218 positions.. correct!
    //Utils::readFENString("r1b1kbnr/pppp1ppp/2n1p3/6q1/6Q1/2N1P3/PPPP1PPP/R1B1KBNR w KQkq - 4 4", &testBoard); // temp test

    int minDepth = 3;
    int maxDepth = 3;
    char fen[1024];
    if (argc >= 3)
    {
        strcpy(fen, argv[1]);
        maxDepth = atoi(argv[2]);
    }
    else
    {
        printf("\nUsage perft_gpu <fen> <depth> [<launchdepth>]\n");
        printf("\nAs no paramaters were provided... running default test\n");
    }

    if (strlen(fen) > 5)
    {
        Utils::readFENString(fen, &testBoard);
    }
    else
    {
        Utils::readFENString("rnbqkbnr/pppppppp/8/8/8/8/PPPPPPPP/RNBQKBNR w KQkq - 0 1", &testBoard); // start.. 20 positions
    }
    Utils::dispBoard(&testBoard);

    

    HexaBitBoardPosition testBB;
    Utils::board088ToHexBB(&testBB, &testBoard);
    Utils::boardHexBBTo088(&testBoard, &testBB);

    // launchDepth is the depth at which the driver kernel launches the work kernels
    // we decide launch depth based by estimating memory requirment of the work kernel that would be launched.

    // TODO: need more accurate method to estimate launch depth
    // branching factor near the root is not accurate. E.g, for start pos, at root branching factor = 20
    // and we estimate launch depth = 6.. which would seem quite conservative (20^6 = 64M)
    // at depth 10, the avg branching factor is nearly 30 and 30^6 = 729M which is > 10X initial estimate :-/
    
    // At launch depth 6, some launches for perft 9 start using up > 350 MB memory
    // 384 MB is not sufficient for computing perft 10 (some of the launches consume more than that)
    // and 1 GB is not sufficient for computing perft 11!
    
    uint32 launchDepth = estimateLaunchDepth(&testBB);
    launchDepth = min(launchDepth, 11); // don't go too high

    // for best performance without GPU hash (also set PREALLOCATED_MEMORY_SIZE to 3 x 768MB)
    // launchDepth = 6;    // ankan - test!

    if (argc >= 4)
    {
        launchDepth = atoi(argv[3]);
    }

    if (maxDepth < launchDepth)
    {
        launchDepth = maxDepth;
    }

    for (int depth = minDepth; depth <= maxDepth; depth++)
    {
        dividedPerft(&testBB, depth, launchDepth);
    }
    
    hipFree(preAllocatedBufferHost);
    hipDeviceReset();
    return 0;
}
