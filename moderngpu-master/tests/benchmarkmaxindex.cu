#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2013, NVIDIA CORPORATION.  All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" 
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE 
 * ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "kernels/reduce.cuh"
#include "kernels/scan.cuh"

#include <thrust/scan.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

using namespace mgpu;



const int Tests[][2] = { 
	{ 10000, 1000 },
	{ 50000, 1000 },
	{ 100000, 1000 },
	{ 200000, 500 },
	{ 500000, 200 },
	{ 1000000, 200 },
	{ 2000000, 200 },
	{ 5000000, 200 },
	{ 10000000, 100 },
	{ 20000000, 100 },
	{ 50000000, 100 }
};
const int NumTests = sizeof(Tests) / sizeof(*Tests);

int main(int argc, char** argv) {
	ContextPtr context = CreateCudaDevice(argc, argv, true);

	typedef int T1;
	printf("Benchmarking max-index on type %s\n", TypeIdName<T1>());
	
	for(int test = 0; test < NumTests; ++test)
		BenchmarkMaxIndex<T1>(Tests[test][0], Tests[test][1], *context);

	typedef int64 T2;
	printf("Benchmarking max-index on type %s\n", TypeIdName<T2>());
	
	for(int test = 0; test < NumTests; ++test)
		BenchmarkMaxIndex<T2>(Tests[test][0], Tests[test][1], *context);

	return 0;
} 
